/**
 * Vector reverse: A[i] = B[SIZE - i].
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

// SIZE is defined to be multiple of the number of threads
#define SIZE 4
#define THREADS_PER_BLOCK 2


__global__ void mat_mul( int *A, int *B, int *C, int size)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tmp_sum = 0;
    int k;

    if (row < size && col < size) {
    for (k = 0; k < size; k++)
    {
    tmp_sum  += A[row * size + k] * B[k * size + col];
    }
    //row+= gridDim.x * blockDim.x;
  }
    C[row * size + col] = tmp_sum;
}



int main( int argc, char * argv[])
{

  int i, j;
  int size_in_bytes = SIZE * SIZE * sizeof(int);
  int *dev_A;
  int *dev_B;
  int *dev_C;

  // Allocate the host input vector A
  int *host_A = (int *) malloc( size_in_bytes );
  int *host_B = (int *) malloc( size_in_bytes );
  int *host_C = (int *) malloc( size_in_bytes );

  // Initialize the host input vectors
  for( i = 0; i < SIZE * SIZE; i++ ){

      host_A[ i ] = i;
      host_B[ i ] = i;
      host_C[ i ] = 0;
  }

  for( i = 0; i < SIZE; i++ )
  {
    for( j = 0; j < SIZE; j++ )
      printf("%d \t", host_A[ i*SIZE + j ]);

    printf("\n");
  }

  for( i = 0; i < SIZE; i++ )
  {
    for( j = 0; j < SIZE; j++ )
      printf("%d \t", host_B[ i*SIZE + j ]);

    printf("\n");
  }

  for( i = 0; i < SIZE; i++ )
  {
    for( j = 0; j < SIZE; j++ )
      printf("%d \t", host_C[ i*SIZE + j ]);

    printf("\n");
  }


  hipMalloc((void **)&dev_A, size_in_bytes);
  hipMalloc((void **)&dev_B, size_in_bytes);
  hipMalloc((void **)&dev_C, size_in_bytes);



  // copy inputs to device
  hipMemcpy(dev_A, host_A, size_in_bytes, hipMemcpyHostToDevice);
  hipMemcpy(dev_B, host_B, size_in_bytes, hipMemcpyHostToDevice);

  int threadsPerBlock = THREADS_PER_BLOCK;
  int blocksPerGrid =( SIZE * SIZE ) / threadsPerBlock;
  mat_mul<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C, SIZE);

  hipMemcpy(host_C, dev_C, size_in_bytes, hipMemcpyDeviceToHost);

  for( i = 0; i < SIZE; i++ )
  {
    for( j = 0; j < SIZE; j++ )
      printf("%d \t", host_C[ i*SIZE + j ]);

    printf("\n");
  }

  printf("-------------------------\n\n");


  hipFree(dev_A);
  hipFree(dev_B);
  hipFree(dev_C);

  // Free host memory

  free(host_A);
  free(host_B);
  free(host_C);

  return 0;
}
