#include "hip/hip_runtime.h"
/**
 * Vector reverse: A[i] = B[SIZE - i].
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

// SIZE is defined to be multiple of the number of threads
#define SIZE 8
#define THREADS_PER_BLOCK 2


__global__ void vectorRev( float *A, float *B)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    A[ index ] = B[ SIZE - index ];
}



int main( int argc, char * argv[])
{

  int i;
  size_t size_in_bytes = SIZE * sizeof(int);

  // Allocate the host input vector A
  int *host_A = (int *) malloc( size_in_bytes );
  int *host_B = (int *) malloc( size_in_bytes );

  int *dev_A = hipMalloc((void **)&dev_A, size_in_bytes);
  int *dev_B = hipMalloc((void **)&dev_B, size_in_bytes);

  // Initialize the host input vectors
  for( i = 0; i < SIZE; i++ ){

      host_A[ i ] = i;
      host_B[ i ] = 0;
  }

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_A[ i ]);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_B[ i ]);


  // copy inputs to device
  hipMemcpy(dev_A, host_A, size_in_bytes, hipMemcpyHostToDevice);
  hipMemcpy(dev_B, host_B, size_in_bytes, hipMemcpyHostToDevice);

  int threadsPerBlock = THREADS_PER_BLOCK;
  int blocksPerGrid =( SIZE ) / threadsPerBlock;
  vectorRev<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B);

  hipMemcpy(host_B, dev_B, size_in_bytes, hipMemcpyDeviceToHost);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_A[ i ]);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_B[ i ]);


  hipFree(dev_A);
  hipFree(dev_B);

  Free(host_A);
  Free(host_B);

  printf("Done\n");
  return 0;
}
