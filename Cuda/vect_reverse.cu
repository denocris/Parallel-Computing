/**
 * Vector reverse: A[i] = B[SIZE - i].
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

// SIZE is defined to be multiple of the number of threads
#define SIZE 8
#define THREADS_PER_BLOCK 2


__global__ void vectorRev( int *A, int *B, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    A[ index ] = B[ size - index ];
}



int main( int argc, char * argv[])
{

  int i;
  int size_in_bytes = SIZE * sizeof(int);
  int *dev_A;
  int *dev_B;

  // Allocate the host input vector A
  int *host_A = (int *) malloc( size_in_bytes );
  int *host_B = (int *) malloc( size_in_bytes );

  // Initialize the host input vectors
  for( i = 0; i < SIZE; i++ ){

      host_A[ i ] = i;
      host_B[ i ] = 0;
  }

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_A[ i ]);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_B[ i ]);

  hipMalloc((void **)&dev_A, size_in_bytes);
  hipMalloc((void **)&dev_B, size_in_bytes);


  // copy inputs to device
  hipMemcpy(dev_A, host_A, size_in_bytes, hipMemcpyHostToDevice);
  //cudaMemcpy(dev_B, host_B, size_in_bytes, cudaMemcpyHostToDevice);

  int threadsPerBlock = THREADS_PER_BLOCK;
  int blocksPerGrid =( SIZE ) / threadsPerBlock;
  vectorRev<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, SIZE);

  hipMemcpy(host_B, dev_B, size_in_bytes, hipMemcpyDeviceToHost);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_A[ i ]);

  for( i = 0; i < SIZE; ++i )
    printf("%d", host_B[ i ]);


  hipFree(dev_A);
  hipFree(dev_B);

  // Free host memory

  free(host_A);
  free(host_B);

  printf("Done\n");
  return 0;
}
