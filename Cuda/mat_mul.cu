#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#define SIZE 2048
#define THREADS_PER_BLOCK 512

// A Simple timer for measuring the walltime
double seconds(){

    struct timeval tmp;
    double sec;
    gettimeofday( &tmp, (struct timezone *)0 );
    sec = tmp.tv_sec + ((double)tmp.tv_usec)/1000000.0;
    return sec;
}

// Multithreaded naive version of Matrix GEMM
void matmul_cpu( double *A, double *B, double *C ){

    int i, j, k;
    double loc_sum;

#pragma omp parallel for private( i, j, k, loc_sum )
    for(i = 0; i < SIZE; i++ ){
        for( j = 0; j < SIZE; j++ ){

	  loc_sum = 0.0;
	  for( k = 0; k < SIZE; k++ ){
	    loc_sum += A[ (SIZE * i) + k ] * B[ (SIZE * k) + j ];
	  }
	  C[ (SIZE * i) + j ] = loc_sum;
        }
    }
}

// GPU naive version of Matrix GEMM using shared memory
__global__ void CUDA_matmul_shared( double *A, double *B, double *C ){

  // Implement e version with shared memory

}


// GPU naive version of Matrix GEMM
__global__ void CUDA_matmul( double *A, double *B, double *C ){

  int i, j, k;
  double loc_sum;
  
  i = blockIdx.x;
  j = threadIdx.x;

  if( i < SIZE ){

    while( j < SIZE){ 
     
      loc_sum = 0.0;
      for( k = 0; k < SIZE; k++ ) loc_sum += A[ (SIZE * i) + k] * B[ ( SIZE * k ) + j];
      C[ (SIZE * i) + j ] = loc_sum;
      j += blockDim.x;
    }
  }
}

/* do matrix multiplication on the GPU here */
void matmul_gpu(double *A, double *B, double *C ){

    size_t size_in_bytes;
    hipError_t err;
    double *d_A, *d_B, *d_C;

    size_in_bytes = sizeof(double) * SIZE * SIZE;
    hipMalloc( (void **) &d_A, size_in_bytes);
    hipMemcpy( d_A, A, size_in_bytes, hipMemcpyHostToDevice );

    hipMalloc( (void **) &d_B, size_in_bytes);
    hipMemcpy( d_B, B, size_in_bytes, hipMemcpyHostToDevice );

    hipMalloc((void **) &d_C, size_in_bytes);

#ifdef __SHARED
    CUDA_matmul_shared<<< SIZE, THREADS_PER_BLOCK >>>( d_A, d_B, d_C );
#else
    CUDA_matmul<<< SIZE, THREADS_PER_BLOCK >>>( d_A, d_B, d_C );
#endif

    /* check if the kernel launch was successful */
    err = hipGetLastError();
    if( err != hipSuccess ){
        printf( "failed to lauch GPU kernel:\n%s\n", hipGetErrorString(err) );
        return;
    }

    hipMemcpy( C, d_C, size_in_bytes, hipMemcpyDeviceToHost );
                                                                             
    hipFree( d_A );
    hipFree( d_B );
    hipFree( d_C );

    hipDeviceReset();    
  
}


int main(int argc, char **argv){

  int flag = 0;

  double time, time_cpu, time_gpu;
  double *A, *B, *C, *C_GPU;
  size_t size_in_bytes;
  int i = 0;
  
  size_in_bytes = SIZE* SIZE * sizeof(double);
  
  A = (double *) malloc( size_in_bytes );
  B = (double *) malloc( size_in_bytes );
  C = (double *) malloc( size_in_bytes );
  C_GPU = (double *) malloc( size_in_bytes );
  
  /* fill matrix A & B on cpu */
#pragma omp parallel for private( i )
  for( i = 0; i < SIZE * SIZE; i++ ){
    
    A[i] = rand() / (double) RAND_MAX;
    B[i] = rand() / (double) RAND_MAX;
  }
  
  time = seconds();
  matmul_cpu( A, B , C );
  time_cpu = seconds() - time;
  
  time = seconds();
  matmul_gpu( A, B, C_GPU );
  time_gpu = seconds() - time;
  

  /* check result */
#ifdef __DEBUG

#pragma omp parallel for private( i ) reduction( +:flag )   
  for( i = 0; i < SIZE * SIZE; i++ ) 
    if( fabs( ( C[i] - C_GPU[i] ) / C[i] ) > 0.000001){
      fprintf( stdout, "\nC[%d] = %.3g\t%.3g", i, C[i], C_GPU[i] );
      flag += 1;
    }
#endif
  
  if( !flag ){

    fprintf( stdout, "Program completed successfully!" );
    fprintf( stdout, "Time for CPU code: %g seconds\n", time_cpu );
    fprintf( stdout, "Time for GPU code: %g seconds\n", time_gpu ); 
  }
  else{
    fprintf( stdout, "Program completed unsuccessfully!" );
  }  

  return 0;
}
